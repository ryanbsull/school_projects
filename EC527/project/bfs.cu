#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include "cuPrintf.cuh"
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <math.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans)                           \
    {                                                 \
        gpuAssert((ans), (char *)__FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define NODES		32768
#define THREADS		8

typedef struct {
  long int *data; 
  long int len;
  long int size;
} queue;

int push(queue *q, long int val){
  if(q->size == q->len-1)
    return 0;
  q->data[q->size++] = val;
  return 1;
}

long int pop(queue *q){
  if(q->size == 0)
    return -1;
  long int val = q->data[0];
  long int i = 0;
  for(; i < q->size; i++)
    q->data[i] = q->data[i+1];
  q->size--;
  return val;
}

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

int get_child_right(int parent){
  return 2*parent + 1;
}

int get_child_left(int parent){
  return 2*parent + 2;
}

void init_tree(long int *t, long int len){
  long int i = 0;
  for(; i < len; i++)
    t[i] = (long int) random();
}

int bfs_serial(long int *tree, long int val, long int len){
  int visited[NODES];
  long int node = 0;
  visited[node] = 1;
  queue *q = (queue*)malloc(sizeof(queue));
  q->len = len; q->size = 0; q->data = (long int *)malloc(sizeof(long int)*len);
  
  push(q, get_child_left(node));
  push(q, get_child_right(node));

  int ret = 0;

  while(q->size > 0){
    node = pop(q);
    // printf("CHECKING NODE: %d\n", node);
    if(visited[node] == 1)
      continue;
    visited[node] = 1;
    if(tree[node] == val){
      ret = 1;
      goto done;
    }
    int l = get_child_left(node), r = get_child_right(node);
    if(l < len)
      push(q, l);
    if(r < len)
      push(q, r);
  }

done:
  free(q);
  return ret;
}

__global__ void kernel_bfs(long int *tree, long int val, long int len, int *detected){
  int visited[NODES];
  int blk_y = (len / gridDim.y);
  int blk_x = (blk_y / gridDim.x);
  int blk_pos = (blk_y * blockIdx.y) + (blk_x * blockIdx.x);
  int t_blk = (blk_x)/(blockDim.x);
  int n = blk_pos + (threadIdx.x*t_blk + (threadIdx.y * (t_blk/blockDim.y)));
  int q[NODES];
  int q_len = 0, q_head = 0, q_tail = 0, cycles = 0;
 
  // cuPrintf("THREAD STARTING AT NODE : \t%d\tNODES PER THREAD: \t%d\n", n, t_blk);

  if(n < len)
    visited[n] = 1;
  else
    return;  

  if(tree[n] == val){
    *detected = 1;
    return;
  }
  
  int l_child = 2*n + 2;
  int r_child = 2*n + 1;
  
  if(l_child < len && !visited[l_child]){
    q[q_tail] = l_child;
    q_tail = (q_tail + 1) % blk_x;
    q_len++; 
  }

  if(r_child < len && !visited[r_child]){
    q[q_tail] = r_child;
    q_tail = (q_tail + 1) % blk_x;
    q_len++; 
  }
  
  while(q_len > 0){
    cycles++;
    n = q[q_head++];
    q_len--;
    
    // cuPrintf("CHECKING NODE : %d\n", n);
    if(tree[n] == val){
      // cuPrintf("%d DETECTED AT NODE : %d\n", val, n);
      *detected = 1;
       break;
    }

    if(visited[n])
      continue;
    
    visited[n] = 1;
    
    l_child = 2*n + 2;
    r_child = 2*n + 1;

    if(l_child < len && !visited[l_child]){ //&& cycles < t_blk){
      q[q_tail] = l_child;
      q_tail = (q_tail + 1) % blk_x;
      q_len++; 
    } 

    if(r_child < len && !visited[r_child]){// && cycles < t_blk){
      q[q_tail] = r_child;
      q_tail = (q_tail + 1) % blk_x;
      q_len++; 
    }
    
  }
}

int main(){
  hipEvent_t start, stop;
  float elapsed_gpu;
  struct timespec time_start, time_stop;
  long int *d_tree;
  long int *tree;
  int detected = 0;
  int *det;
  float measurement;
  size_t allocSize = sizeof(long int) * NODES;

  printf("TREE SIZE: \t%d NODES\n", NODES);

  CUDA_SAFE_CALL(hipSetDevice(0));
  
  tree = (long int*)malloc(allocSize);

  CUDA_SAFE_CALL(hipMalloc((void **)&d_tree, allocSize));
  CUDA_SAFE_CALL(hipMalloc((void **)&det, sizeof(int)));
  
  printf("\nInitializing tree ...");
  init_tree(tree, NODES);
  printf("\t... done\n");

/* 
  int i = 0;
  for(; i < NODES; i++)
    tree[i] = i;
*/

  long int val = tree[NODES-1];


  CUDA_SAFE_CALL(hipMemcpy(d_tree, tree, allocSize, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(det, &detected, sizeof(int), hipMemcpyHostToDevice));

  // init CUDA printf
  // cudaPrintfInit();

  int blk = sqrt(NODES / (THREADS * THREADS));
  printf("BLOCKS: %d X %d\n", blk, blk);
  dim3 threads(THREADS, THREADS);
  dim3 blocks(blk,blk);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
 
  kernel_bfs<<<blocks, threads>>>(d_tree, val, NODES, det);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);

  CUDA_SAFE_CALL(hipPeekAtLastError());

  // print from CUDA
  // cudaPrintfDisplay(stdout, true); cudaPrintfEnd();

  CUDA_SAFE_CALL(hipMemcpy(&detected, det, sizeof(int), hipMemcpyDeviceToHost));

  printf("\nGPU time: %f (msec)\n", elapsed_gpu);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  clock_gettime(CLOCK_REALTIME, &time_start);
  bfs_serial(tree, val, NODES);
  clock_gettime(CLOCK_REALTIME, &time_stop);
  measurement = interval(time_start, time_stop);

  printf("\nSERIAL time: %f (msec)\n", 1000*measurement);

  return 0;
}
