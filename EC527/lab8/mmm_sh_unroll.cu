#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include "cuPrintf.cuh"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans)                           \
    {                                                 \
        gpuAssert((ans), (char *)__FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define NUM_THREADS_2D 16
#define NUM_BLOCKS_2D 16
#define PRINT_TIME 1
#define ROW_LEN 2048
#define TILE_WIDTH 16
#define TOL 0.0001

#define IMUL(a, b) __mul24(a, b)

void initArr2D(float *arr, int rowlen, int seed)
{
    int i;
    float randNum;
    srand(seed);

    for (i = 0; i < rowlen * rowlen; i++)
    {
        randNum = (float)rand();
        arr[i] = randNum;
    }
}

void cpyArr(float *src, float *dest, int rowlen)
{
    int i;
    for (i = 0; i < rowlen * rowlen; i++)
    {
        dest[i] = src[i];
    }
}

float interval(struct timespec start, struct timespec end)
{
    struct timespec temp;
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    if (temp.tv_nsec < 0)
    {
        temp.tv_sec = temp.tv_sec - 1;
        temp.tv_nsec = temp.tv_nsec + 1000000000;
    }
    return (((float)temp.tv_sec) + ((float)temp.tv_nsec) * 1.0e-9);
}

__global__ void kernel_mmm(int rowlen, float *d_x, float *d_y, float *d_out)
{
    __shared__ float blk_x[TILE_WIDTH][TILE_WIDTH];
    __shared__ float blk_y[TILE_WIDTH][TILE_WIDTH];
    int row = (blockIdx.y * TILE_WIDTH) + threadIdx.y;
    int col = (blockIdx.x * TILE_WIDTH) + threadIdx.x;
    float val0, val1, val2, val3, val4, val5;

    int i, k, tx = threadIdx.x, ty = threadIdx.y;
    val0 = 0;
    val1 = 0;
    val2 = 0;
    val3 = 0;
    val4 = 0;
    val5 = 0;

    for (i = 0; i < rowlen / TILE_WIDTH; i++)
    {
        blk_x[ty][tx] = d_x[row * rowlen + (i * TILE_WIDTH + tx)];
        blk_y[ty][tx] = d_y[col + ((i * TILE_WIDTH + ty) * rowlen)];
        __syncthreads();

        for (k = 0; k < TILE_WIDTH-6; k+=6){
          val0 += blk_x[ty][k] * blk_y[k][tx];
          val1 += blk_x[ty][k+1] * blk_y[k+1][tx];
          val2 += blk_x[ty][k+2] * blk_y[k+2][tx];
          val3 += blk_x[ty][k+3] * blk_y[k+3][tx];
          val4 += blk_x[ty][k+4] * blk_y[k+4][tx];
          val5 += blk_x[ty][k+5] * blk_y[k+5][tx];
        }
        for(; k < TILE_WIDTH; k++)
          val0 += blk_x[ty][k] * blk_y[k][tx];
        __syncthreads();
    }
    d_out[row * rowlen + col] = (val0 + (val1 + (val2 + (val3 + (val4 + val5)))));
}

int main(int argc, char **argv)
{
    int rowlen = ROW_LEN;

    // GPU Timing variables
    hipEvent_t start, stop;
    float elapsed_gpu;

    // CPU Timing variable
    struct timespec time_start, time_stop;
    float time_stamp;

    // Arrays on GPU global memory
    float *d_x;
    float *d_y;
    float *d_out;

    // Arrays on host memory
    float *x;
    float *y;
    float *out;
    float *check;

    int i, j, k, errCount = 0, zeroCount = 0;

    printf("Size of the matrix = %d X %d\n", rowlen, rowlen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // Set allocation size
    size_t allocSize = rowlen * rowlen * sizeof(float);

    // Allocate GPU memory
    CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_out, allocSize));

    // Allocate arrays on host memory
    x = (float *)malloc(allocSize);
    y = (float *)malloc(allocSize);
    out = (float *)malloc(allocSize);
    check = (float *)malloc(allocSize);

    // Initialize the host arrays
    printf("\nInitializing the arrays ...");
    // Arrays are initialized with a known seed for reproducability
    initArr2D(x, rowlen, 1065);
    initArr2D(y, rowlen, 1016);

    printf("\t... done\n");

#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    CUDA_SAFE_CALL(hipMemcpy(d_x, x, allocSize, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_y, y, allocSize, hipMemcpyHostToDevice));

    // init CUDA printf
    // cudaPrintfInit();

    // Setup thread structure
    dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 blocks(ROW_LEN / TILE_WIDTH, ROW_LEN / TILE_WIDTH);

    // Launch the kernel
    kernel_mmm<<<blocks, threads>>>(rowlen, d_x, d_y, d_out);

    // print from CUDA
    // cudaPrintfDisplay(stdout, true);
    // cudaPrintfEnd();

    // Check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    // Transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(out, d_out, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

    // Compute the results on the host
    float r;
    clock_gettime(CLOCK_REALTIME, &time_start);
    for (k = 0; k < rowlen; k++)
    {
        for (i = 0; i < rowlen; i++)
        {
            r = x[i * rowlen + k];
            for (j = 0; j < rowlen; j++)
            {
                check[i * rowlen + j] += r * y[k * rowlen + j];
            }
        }
    }
    clock_gettime(CLOCK_REALTIME, &time_stop);
    time_stamp = interval(time_start, time_stop);
    printf("\nCPU time: %f (msec)\n", 1000 * time_stamp);
    // Compare the results

    for (i = 0; i < rowlen; i++)
    {
        for (j = 0; j < rowlen; j++)
        {

            float result = abs((check[i * rowlen + j] - out[i * rowlen + j]) / (out[i * rowlen + j]));
            if (result > TOL)
            {
                errCount++;
            }
            if (out[i * rowlen + j] == 0)
            {
                zeroCount++;
            }
        }
    }
    printf("\033[0m");

    if (errCount > 0)
    {
        printf("\n@ERROR: TEST FAILED: %d results did not match\n", errCount);
    }
    if (zeroCount > 0)
    {
        printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
    }
    if (!errCount && !zeroCount)
    {
        printf("\nTEST PASSED: All results matched\n");
    }

    // Free-up device and host memory
    CUDA_SAFE_CALL(hipFree(d_x));
    CUDA_SAFE_CALL(hipFree(d_y));
    CUDA_SAFE_CALL(hipFree(d_out));

    free(x);
    free(y);
    free(out);

    return 0;
}

