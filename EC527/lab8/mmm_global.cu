#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include "cuPrintf.cuh"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans)                           \
    {                                                 \
        gpuAssert((ans), (char *)__FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define NUM_THREADS_2D 16
#define NUM_BLOCKS_2D 16
#define PRINT_TIME 1
#define ROW_LEN 1024
#define TOL 0.0001

#define IMUL(a, b) __mul24(a, b)

void initArr2D(float *arr, int rowlen, int seed)
{
    int i;
    float randNum;
    srand(seed);

    for (i = 0; i < rowlen * rowlen; i++)
    {
        randNum = (float)rand();
        arr[i] = randNum;
    }
}

void cpyArr(float *src, float *dest, int rowlen)
{
    int i;
    for (i = 0; i < rowlen * rowlen; i++)
    {
        dest[i] = src[i];
    }
}

float interval(struct timespec start, struct timespec end)
{
    struct timespec temp;
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    if (temp.tv_nsec < 0)
    {
        temp.tv_sec = temp.tv_sec - 1;
        temp.tv_nsec = temp.tv_nsec + 1000000000;
    }
    return (((float)temp.tv_sec) + ((float)temp.tv_nsec) * 1.0e-9);
}

__global__ void kernel_mmm(int rowlen, float *d_x, float *d_y, float *d_out) {
  int blk_sz = rowlen / gridDim.x;
  int t_sz = blk_sz / blockDim.x;
  int row = t_sz*(blockIdx.y*blockDim.y + threadIdx.y);
  int col = t_sz*(blockIdx.x*blockDim.x + threadIdx.x);
  float val;
  int i,j,k;

  for(i = row; i < row+t_sz; i++){
    for(j = col; j < col+t_sz; j++){
      val = 0;
      if(i < rowlen && j < rowlen) {
        for(k = 0; k < rowlen; k++) {
          val += d_x[i*rowlen + k] * d_y[k*rowlen + j];
        }
        d_out[i*rowlen + j] = val;
      }
    }
  }
}

int main(int argc, char **argv)
{
    int rowlen = ROW_LEN;

    // GPU Timing variables
    hipEvent_t start, stop;
    float elapsed_gpu;

    // CPU Timing variable
    struct timespec time_start, time_stop;
    float time_stamp;

    // Arrays on GPU global memory
    float *d_x;
    float *d_y;
    float *d_out;

    // Arrays on host memory
    float *x;
    float *y;
    float *out;
    float *check;

    int i, j, k, errCount = 0, zeroCount = 0;

    printf("Size of the matrix = %d X %d\n", rowlen, rowlen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // Set allocation size
    size_t allocSize = rowlen * rowlen * sizeof(float);

    // Allocate GPU memory
    CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_out, allocSize));

    // Allocate arrays on host memory
    x = (float *)malloc(allocSize);
    y = (float *)malloc(allocSize);
    out = (float *)malloc(allocSize);
    check = (float *)malloc(allocSize);

    // Initialize the host arrays
    printf("\nInitializing the arrays ...");
    // Arrays are initialized with a known seed for reproducability
    initArr2D(x, rowlen, 1065);
    initArr2D(y, rowlen, 1016);

    printf("\t... done\n");

#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    CUDA_SAFE_CALL(hipMemcpy(d_x, x, allocSize, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_y, y, allocSize, hipMemcpyHostToDevice));

    // init CUDA printf
    // cudaPrintfInit();

    // Setup thread structure
    dim3 threads(NUM_THREADS_2D, NUM_THREADS_2D);
    dim3 blocks(NUM_BLOCKS_2D, NUM_BLOCKS_2D);

    // Launch the kernel
    kernel_mmm<<<blocks, threads>>>(rowlen, d_x, d_y, d_out);

    // print from CUDA
    // cudaPrintfDisplay(stdout, true); cudaPrintfEnd();

    // Check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    // Transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(out, d_out, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

    // Compute the results on the host
    float r;
    clock_gettime(CLOCK_REALTIME, &time_start);
    for (k = 0; k < rowlen; k++)
    {
        for (i = 0; i < rowlen; i++)
        {
            r = x[i * rowlen + k];
            for (j = 0; j < rowlen; j++)
            {
                check[i * rowlen + j] += r * y[k * rowlen + j];
	    }
        }
    }
    clock_gettime(CLOCK_REALTIME, &time_stop);
    time_stamp = interval(time_start, time_stop);
    printf("\nCPU time: %f (msec)\n", 1000 * time_stamp);
    // Compare the results

    for (i = 0; i < rowlen; i++) {
        for (j = 0; j < rowlen; j++) {

            float result = abs((check[i * rowlen + j] - out[i * rowlen + j]) / (out[i * rowlen + j]));
            if (result > TOL) {
                errCount++;
            }
            if (out[i * rowlen + j] == 0){
                zeroCount++;
            }
        }
    }
    printf("\033[0m");

    if (errCount > 0){
        printf("\n@ERROR: TEST FAILED: %d results did not match\n", errCount);
    }
    if (zeroCount > 0){
        printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
    }
    if (!errCount && !zeroCount){
        printf("\nTEST PASSED: All results matched\n");
    }

    // Free-up device and host memory
    CUDA_SAFE_CALL(hipFree(d_x));
    CUDA_SAFE_CALL(hipFree(d_y));
    CUDA_SAFE_CALL(hipFree(d_out));

    free(x);
    free(y);
    free(out);

    return 0;
}
