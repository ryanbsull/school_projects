#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include "cuPrintf.cuh"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define NUM_THREADS_2D	16
#define NUM_BLOCKS_2D	16
#define PRINT_TIME	1
#define ROW_LEN		2048
#define	TOL		0.05
#define OMEGA		1.68
#define ITERATIONS	2000

#define IMUL(a, b) __mul24(a, b)

void initArr2D(float *arr, int rowlen, int seed){
  int i;
  float randNum;
  srand(seed);

  for (i = 0; i < rowlen*rowlen; i++) {
    randNum = (float) rand();
    arr[i] = randNum;
  }
}

void cpyArr(float *src, float *dest, int rowlen){
  int i;
  for (i = 0; i < rowlen*rowlen; i++) {
    dest[i] = src[i];
  }
}

float interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((float)temp.tv_sec) + ((float)temp.tv_nsec)*1.0e-9);
}

__global__ void kernel_sor(int rowlen, float* d_x){
  const int blk_sz 	= (int) (rowlen / gridDim.x);
  const int blk_idx 	= blk_sz * blockIdx.x;
  const int blk_off 	= blk_sz * blockIdx.y;
  const int t_sz 	= (int) (blk_sz / blockDim.x);
  const int tidx	= blk_idx + t_sz * threadIdx.x;
  const int toff	= blk_off + t_sz * threadIdx.y;

  int i, j;
  double change;

  for(i = tidx; i < tidx+t_sz; i++) { 
    for(j = toff; j < toff+t_sz; j++) {
      if(i > 0 && j > 0 && i < rowlen-1 && j < rowlen-1) {
        change = d_x[i*rowlen+j] - .25 * (d_x[(i-1)*rowlen+j] +
                                          d_x[(i+1)*rowlen+j] +
                                          d_x[i*rowlen+j+1] +
                                          d_x[i*rowlen+j-1]);
        d_x[i*rowlen+j] -= change * OMEGA;
      }
    }
  }
}

int main(int argc, char **argv){
  int rowlen = ROW_LEN;

  // GPU Timing variables
  hipEvent_t start, stop;
  float elapsed_gpu;

  // CPU Timing variable
  struct timespec time_start, time_stop;
  float time_stamp;

  // Arrays on GPU global memoryc
  float *d_x;
  
  // Arrays on host memory
  float *x;
  float *x_cpy;

  int i, j, it, errCount = 0, zeroCount = 0;

  printf("Length of the array = %d\n", rowlen*rowlen);

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));

  // Allocate GPU memory
  size_t allocSize = rowlen * rowlen * sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));

  // Allocate arrays on host memory
  x                        = (float *) malloc(allocSize);
  x_cpy                    = (float *) malloc(allocSize);

  // Initialize the host arrays
  printf("\nInitializing the arrays ...");
  // Arrays are initialized with a known seed for reproducability
  initArr2D(x, rowlen, 2453);
  initArr2D(x_cpy, rowlen, 2453);
  //cpyArr(x, x_cpy, rowlen);
  for(i = 0; i < rowlen*rowlen; i++) {
    if (abs(x_cpy[i] - x[i]) > TOL) {
      printf("@ERROR: INACCURATE COPY\n");
      return -1; 
    }
  }
  printf("\t... done\n");

#if PRINT_TIME
  // Create the cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Record event on the default stream
  hipEventRecord(start, 0);
#endif

  // Transfer the arrays to the GPU memory
  CUDA_SAFE_CALL(hipMemcpy(d_x, x, allocSize, hipMemcpyHostToDevice));
  
  // init CUDA printf
  // cudaPrintfInit();
  
  // Setup thread structure
  dim3 threads(NUM_THREADS_2D, NUM_THREADS_2D);
  dim3 blocks(NUM_BLOCKS_2D, NUM_BLOCKS_2D);

  // Launch the kernel
  for(it = 0; it < ITERATIONS; it++)
    kernel_sor<<<blocks, threads>>>(rowlen, d_x);

  // print from CUDA
  // cudaPrintfDisplay(stdout, true); cudaPrintfEnd();

  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());

  // Transfer the results back to the host
  CUDA_SAFE_CALL(hipMemcpy(x, d_x, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
  // Stop and destroy the timer
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);
  printf("\nGPU time: %f (msec)\n", elapsed_gpu);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif
  
  float change;
  // Compute the results on the host
  clock_gettime(CLOCK_REALTIME, &time_start);
  for(it = 0; it < ITERATIONS; it++){
    for(i = 1; i < rowlen-1; i++){
      for(j = 1; j < rowlen-1; j++){
        change = x_cpy[i*rowlen+j] - .25 * (x_cpy[(i-1)*rowlen+j] +
                                            x_cpy[(i+1)*rowlen+j] +
                                            x_cpy[i*rowlen+j+1] +
                                            x_cpy[i*rowlen+j-1]);
        x_cpy[i*rowlen+j] -= change * OMEGA;
      }
    }
  }
  clock_gettime(CLOCK_REALTIME, &time_stop);
  time_stamp = interval(time_start, time_stop);
  printf("\nCPU time: %f (msec)\n", 1000*time_stamp);
  // Compare the results
/*
  for(i = 0; i < rowlen*rowlen; i++) {
    printf("%d:\t%.8f\t%.8f\t%d\n", i, x[i], x_cpy[i], !(x[i] == x_cpy[i]));
  }
*/
  for(i = 0; i < rowlen; i++) {
    for(j = 0; j < rowlen; j++) {
      if (abs(x_cpy[i*rowlen + j] - x[i*rowlen + j])/(x_cpy[i*rowlen + j]) > TOL) {
        errCount++;
      }
      if (x[i*rowlen + j] == 0) {
        zeroCount++;
      }
    }
  }
  printf("\033[0m");

  if (errCount > 0) {
    printf("\n@ERROR: TEST FAILED: %d results did not match\n", errCount);
  }
  if (zeroCount > 0){
    printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
  }
  if(!errCount && !zeroCount) {
    printf("\nTEST PASSED: All results matched\n");
  }

  // Free-up device and host memory
  CUDA_SAFE_CALL(hipFree(d_x));

  free(x);
  free(x_cpy);

  return 0;
}
